
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define BLOCKS  size
#define THREADS 1
#define T       10000
#define H       0.01
#define R       1.0
#define K       1.0
#define ALPHA   1.0
#define BETA    5.0
#define M       0.2
#define DN      0.0
#define DP      0.05

__global__ void rosmac(float *n0, float *n1, float *p0, float *p1)
{
    // Better integration:
    const int tid = blockIdx.x;
    const int left = (tid == 0)? blockDim.x  - 1 : tid - 1;
    const int right = (tid == blockDim.x - 1)? 0 : tid + 1;
    const float dn = R * n0[tid] * (1.0f - n0[tid] / K) - (ALPHA * n0[tid] * p0[tid]) / (1.0f + BETA * n0[tid]) - DN * (n0[tid] - n0[left] / 2.0f - n0[right] / 2.0f);
    const float dp = (ALPHA * n0[tid] * p0[tid]) / (1.0f + BETA * n0[tid]) - M * p0[tid] - DP * (p0[tid] - p0[left] / 2.0f - p0[right] / 2.0f);
    n1[tid] = n0[tid] + H * dn;
    p1[tid] = p0[tid] + H * dp;
}

int main(int argc, char **argv)
{
    const unsigned int size = (argc == 2)? atof(argv[1]) : 1000;
    const unsigned int bytes = size * sizeof(float);
    float *h_n = (float*)malloc(bytes);
    float *h_p = (float*)malloc(bytes);

    float *d_n0, *d_n1, *d_p0, *d_p1;
    hipMalloc((void**)&d_n0, bytes);
    hipMalloc((void**)&d_n1, bytes);
    hipMalloc((void**)&d_p0, bytes);
    hipMalloc((void**)&d_p1, bytes);

    // Use gsl:
    srand(42);
    rand();
    for (int i = 0; i < size; ++i)
    {
        h_n[i] = ((float)rand() / RAND_MAX);
        h_p[i] = ((float)rand() / RAND_MAX);
    }

    hipMemcpy(d_n0, h_n, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_p0, h_p, bytes, hipMemcpyHostToDevice);

    for (int t = 0; t < T; t += 2)
    {
        rosmac<<<BLOCKS,THREADS>>>(d_n0, d_n1, d_p0, d_p1);
        rosmac<<<BLOCKS,THREADS>>>(d_n1, d_n0, d_p1, d_p0);
        if (t % 100 == 0)
        {
            //printf("%16d -> ", t);
            hipMemcpy(h_n, d_n1, bytes, hipMemcpyDeviceToHost);
            hipMemcpy(h_p, d_p1, bytes, hipMemcpyDeviceToHost);
            for (int i = 0; i < size; ++i)
            {
                printf("%.4f\t", h_p[i]);
            }
            printf("\n");
        }
    }

    hipFree(d_n0);
    hipFree(d_n1);
    hipFree(d_p0);
    hipFree(d_p1);
    free(h_n);
    free(h_p);
    return EXIT_SUCCESS;
}

